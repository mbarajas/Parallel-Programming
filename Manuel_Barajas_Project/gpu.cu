#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 8192
#define MEMSIZE 32


__global__ void multiplyTest(double * one, double * two, double * three,
                                     int size) {
    __shared__ double s_first[MEMSIZE][MEMSIZE];
    __shared__ double s_second[MEMSIZE][MEMSIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * MEMSIZE + ty;
    int col = bx * MEMSIZE + tx;

    double elementSum = 0;

    for(int m = 0; m < size / MEMSIZE; m++) {
        s_first[ty][tx] = one[row * size + m * MEMSIZE + tx];
        s_second[ty][tx] = two[(m * MEMSIZE + ty) * size + col];

        __syncthreads();

        for(int k = 0; k < MEMSIZE; k++)
            elementSum += s_first[ty][k] * s_second[k][tx];

        __syncthreads();

    }
    three[row * size + col] = elementSum;
}

int main() {
    int c;
    double *h_first = (double *) malloc(SIZE * SIZE * sizeof(double));
    double *h_second = (double *) malloc(SIZE * SIZE * sizeof(double));
    double *h_sum = (double *) malloc(SIZE * SIZE * sizeof(double));

    long i;
    for(i = 0; i < SIZE * SIZE; i++) {
        h_first[i] = 2.0;
        h_second[i] = 2.0;
        h_sum[i] = 0.0;
    }

    double *d_first = NULL;
    double *d_second = NULL;
    double *d_sum = NULL;

    hipMalloc((void **) &d_first, SIZE * SIZE * sizeof(double));
    hipMalloc((void **) &d_second, SIZE * SIZE * sizeof(double));
    hipMalloc((void **) &d_sum, SIZE * SIZE * sizeof(double));

    hipMemcpy(d_first, h_first, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_second, h_second, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimGrid(SIZE / MEMSIZE, SIZE / MEMSIZE,1);
    dim3 dimBlock(MEMSIZE, MEMSIZE, 1);

    c = clock();

    multiplyTest<<<dimGrid,dimBlock>>>(d_first, d_second, d_sum, SIZE);

    hipDeviceSynchronize();
    hipMemcpy(h_sum, d_sum, SIZE * SIZE * sizeof(double), hipMemcpyDeviceToHost);

    printf("Done. Time: %d\n", clock()-c);

    return 0;
}


